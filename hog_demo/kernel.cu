#include "hip/hip_runtime.h"

#include "kernel.h"

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/warp_shuffle.hpp"

#include <stdio.h>

namespace test
{
	hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

	__global__ void addKernel(int *c, const int *a, const int *b)
	{
		int i = threadIdx.x;
		c[i] = a[i] + b[i];
	}

	int test_main()
	{
		const int arraySize = 5;
		const int a[arraySize] = { 1, 2, 3, 4, 5 };
		const int b[arraySize] = { 10, 20, 30, 40, 50 };
		int c[arraySize] = { 0 };

		// Add vectors in parallel.
		hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
			c[0], c[1], c[2], c[3], c[4]);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}

	// Helper function for using CUDA to add vectors in parallel.
	hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
	{
		int *dev_a = 0;
		int *dev_b = 0;
		int *dev_c = 0;
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		addKernel << <1, size >> >(dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		return cudaStatus;
	}
}


/*�����ڴ�
// λ�ã��豸�ڴ�
// ��ʽ���ؼ���__constant__��ӵ����������С���__constant__ float s[10]; ��
// Ŀ�ģ�Ϊ���������ܡ������ڴ��ȡ�˲�ͬ�ڱ�׼ȫ���ڴ�Ĵ���ʽ����ĳЩ����£��ó����ڴ��滻ȫ���ڴ�����Ч�ؼ����ڴ����
// �ص㣺�����ڴ����ڱ����ں˺���ִ���ڼ䲻�ᷢ���仯�����ݡ������ķ�������Ϊֻ����NVIDIAӲ���ṩ��64KB�ĳ����ڴ档������ҪhipMalloc()����hipFree(), �����ڱ���ʱ����̬�ط���ռ䡣
// Ҫ�󣺵�������Ҫ�������ݵ������ڴ���Ӧ��ʹ��hipMemcpyToSymbol()����hipMemcpy()�Ḵ�Ƶ�ȫ���ڴ档
// ����������ԭ��
    �Գ����ڴ�ĵ��ζ��������Թ㲥�������ġ��ڽ����̡߳��⽫��Լ15�ζ�ȡ��������Ϊʲô��15����Ϊ���ڽ���ָ����߳�����һ���߳�������32���̵߳ļ��ϡ���
    �����ڴ�����ݽ�������������˶���ͬ��ַ���������������������������ڴ�ͨ������
*/

// using CUDA to hog
__constant__ int cnbins;				// ֱ��ͼbin������(ͶƱ��ĸ���)
__constant__ int cblock_stride_x;		// x�����Ļ�����������Сֻ֧���ǵ�Ԫ��cell_size��С�ı���
__constant__ int cblock_stride_y;		//
__constant__ int cnblocks_win_x;		// x���� ÿ��window�е�block��
__constant__ int cnblocks_win_y;		// 
__constant__ int cncells_block_x;		// x���� ÿ��block�е�cell��
__constant__ int cncells_block_y;		//
__constant__ int cblock_hist_size;		// ÿ��block��ֱ��ͼ��С
__constant__ int cblock_hist_size_2up;	// 
__constant__ int cdescr_size;			//HOG����������ά��
__constant__ int cdescr_width;			//


/* ������ӽ����������ޣ���������
���͵�GPU�߳�����pert block��ֵ */
int power_2up(unsigned int n)
{
	if (n <= 1) return 1;
	else if (n <= 2) return 2;
	else if (n <= 4) return 4;
	else if (n <= 8) return 8;
	else if (n <= 16) return 16;
	else if (n <= 32) return 32;
	else if (n <= 64) return 64;
	else if (n <= 128) return 128;
	else if (n <= 256) return 256;
	else if (n <= 512) return 512;
	else if (n <= 1024) return 1024;
	return -1; // Input is too big
}

/* ����nblocks�����ֵ */
int max_nblocks(int nthreads, int ncells_block = 1)
{
	int threads = nthreads * ncells_block;
	if (threads * 4 <= 256)
		return 4;
	else if (threads * 3 <= 256)
		return 3;
	else if (threads * 2 <= 256)
		return 2;
	else
		return 1;
}

/*
// nbins��ֱ��ͼbin��������Ŀǰÿ����Ԫ��Cellֻ֧��9��
// block_stride_x��width����block�Ļ�����������Сֻ֧�ֵ�Ԫ��cell_size��С�ı���
//
// nblocks_win_x��blocks_per_win.width
//
// ncells_block_x��cells_per_block_.width
//
*/
void set_up_constants(int nbins,
	int block_stride_x, int block_stride_y,
	int nblocks_win_x, int nblocks_win_y,
	int ncells_block_x, int ncells_block_y,
	const hipStream_t& stream)
{
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cnbins), &nbins, sizeof(nbins), 0, hipMemcpyHostToDevice, stream));
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cblock_stride_x), &block_stride_x, sizeof(block_stride_x), 0, hipMemcpyHostToDevice, stream));
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cblock_stride_y), &block_stride_y, sizeof(block_stride_y), 0, hipMemcpyHostToDevice, stream));
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cnblocks_win_x), &nblocks_win_x, sizeof(nblocks_win_x), 0, hipMemcpyHostToDevice, stream));
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cnblocks_win_y), &nblocks_win_y, sizeof(nblocks_win_y), 0, hipMemcpyHostToDevice, stream));
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cncells_block_x), &ncells_block_x, sizeof(ncells_block_x), 0, hipMemcpyHostToDevice, stream));
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cncells_block_y), &ncells_block_y, sizeof(ncells_block_y), 0, hipMemcpyHostToDevice, stream));

	int block_hist_size = nbins * ncells_block_x * ncells_block_y;
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cblock_hist_size), &block_hist_size, sizeof(block_hist_size), 0, hipMemcpyHostToDevice, stream));

	//��ӽ������ޣ���gpu per block
	int block_hist_size_2up = power_2up(block_hist_size);
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cblock_hist_size_2up), &block_hist_size_2up, sizeof(block_hist_size_2up), 0, hipMemcpyHostToDevice, stream));

	int descr_width = nblocks_win_x * block_hist_size;
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdescr_width), &descr_width, sizeof(descr_width), 0, hipMemcpyHostToDevice, stream));

	int descr_size = descr_width * nblocks_win_y;
	cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(cdescr_size), &descr_size, sizeof(descr_size), 0, hipMemcpyHostToDevice, stream));
}


//----------------------------------------------------------------------------
// ֱ��ͼ����
//
// CUDA�ں�������ֱ��ͼ
template <int nblocks> // ����GPU�߳̿鴦���ֱ��ͼ�������
__global__ void compute_hists_kernel_many_blocks(const int img_block_width, const cv::cuda::PtrStepf grad,
	const cv::cuda::PtrStepb qangle, float scale, float* block_hists,
	int cell_size, int patch_size, int block_patch_size,
	int threads_cell, int threads_block, int half_cell_size)
{
	const int block_x = threadIdx.z;
	const int cell_x = threadIdx.x / threads_cell;
	const int cell_y = threadIdx.y;
	const int cell_thread_x = threadIdx.x & (threads_cell - 1);

	if (blockIdx.x * blockDim.z + block_x >= img_block_width)
		return;

	extern __shared__ float smem[];
	float* hists = smem;
	float* final_hist = smem + cnbins * block_patch_size * nblocks;

	// patch_size means that patch_size pixels affect on block's cell �����⣿
	if (cell_thread_x < patch_size)
	{
		const int offset_x = (blockIdx.x * blockDim.z + block_x) * cblock_stride_x +
			half_cell_size * cell_x + cell_thread_x;
		const int offset_y = blockIdx.y * cblock_stride_y + half_cell_size * cell_y;

		const float* grad_ptr = grad.ptr(offset_y) + offset_x * 2;
		const unsigned char* qangle_ptr = qangle.ptr(offset_y) + offset_x * 2;


		float* hist = hists + patch_size * (cell_y * blockDim.z * cncells_block_y +
			cell_x + block_x * cncells_block_x) +
			cell_thread_x;
		for (int bin_id = 0; bin_id < cnbins; ++bin_id)
			hist[bin_id * block_patch_size * nblocks] = 0.f;

		//(dist_x, dist_y) : distance between current pixel in patch and cell's center
		const int dist_x = -half_cell_size + (int)cell_thread_x - half_cell_size * cell_x;

		const int dist_y_begin = -half_cell_size - half_cell_size * (int)threadIdx.y;
		for (int dist_y = dist_y_begin; dist_y < dist_y_begin + patch_size; ++dist_y)
		{
			float2 vote = *(const float2*)grad_ptr;
			uchar2 bin = *(const uchar2*)qangle_ptr;

			grad_ptr += grad.step / sizeof(float);
			qangle_ptr += qangle.step;

			//(dist_center_x, dist_center_y) : distance between current pixel in patch and block's center
			int dist_center_y = dist_y - half_cell_size * (1 - 2 * cell_y);
			int dist_center_x = dist_x - half_cell_size * (1 - 2 * cell_x);

			float gaussian = ::expf(-(dist_center_y * dist_center_y +
				dist_center_x * dist_center_x) * scale);

			float interp_weight = ((float)cell_size - ::fabs(dist_y + 0.5f)) *
				((float)cell_size - ::fabs(dist_x + 0.5f)) / (float)threads_block;

			hist[bin.x * block_patch_size * nblocks] += gaussian * interp_weight * vote.x;
			hist[bin.y * block_patch_size * nblocks] += gaussian * interp_weight * vote.y;
		}

		//reduction of the histograms
		volatile float* hist_ = hist;
		for (int bin_id = 0; bin_id < cnbins; ++bin_id, hist_ += block_patch_size * nblocks)
		{
			if (cell_thread_x < patch_size / 2) hist_[0] += hist_[patch_size / 2];
			if (cell_thread_x < patch_size / 4 && (!((patch_size / 4) < 3 && cell_thread_x == 0)))
				hist_[0] += hist_[patch_size / 4];
			if (cell_thread_x == 0)
				final_hist[((cell_x + block_x * cncells_block_x) * cncells_block_y + cell_y) * cnbins + bin_id]
				= hist_[0] + hist_[1] + hist_[2];
		}
	}

	__syncthreads();

	float* block_hist = block_hists + (blockIdx.y * img_block_width +
		blockIdx.x * blockDim.z + block_x) *
		cblock_hist_size;

	//��final_hist���Ƶ�block_hist
	int tid;
	if (threads_cell < cnbins)
	{
		tid = (cell_y * cncells_block_y + cell_x) * cnbins + cell_thread_x;
	}
	else
	{
		tid = (cell_y * cncells_block_y + cell_x) * threads_cell + cell_thread_x;
	}
	if (tid < cblock_hist_size)
	{
		block_hist[tid] = final_hist[block_x * cblock_hist_size + tid];
		if (threads_cell < cnbins && cell_thread_x == (threads_cell - 1))
		{
			for (int i = 1; i <= (cnbins - threads_cell); ++i)
			{
				block_hist[tid + i] = final_hist[block_x * cblock_hist_size + tid + i];
			}
		}
	}
}


/*
// nbins��ֱ��ͼbin��������Ŀǰÿ����Ԫ��Cellֻ֧��9��
// block_stride_x��x�����Ļ�����������Сֻ֧���ǵ�Ԫ��cell_size��С�ı���

// Դͼ��ֻ֧��CV_8UC1��CV_8UC4��������
// height������ͼ������rows
// width������ͼ������cols

// grad������ݶȣ���ͨ��������¼ÿ����������bin��Ӧ��Ȩ�صľ���Ϊ��ֵ����Ȩֵ�����Ȩֵ�ǹؼ���Ҳ�ܸ��ӣ�������˹Ȩ�أ����β�ֵ��Ȩ�أ��ڱ���������ֻ���Ƿ�ֵ������bin��Ĳ�ֵȨ��
// qangle�����뻡�ȣ���ͨ��������¼ÿ�����ؽǶ�������bin��ŵľ���,��Ϊ2ͨ��,Ϊ�����Բ�ֵ
// sigma��winSigma����˹�˲����ڵĲ���
// *block_hists��block_hists.ptr<float>����
*/

// �������������ü����blocks������kernel
void compute_hists(int nbins,
	int block_stride_x, int block_stride_y,
	int height, int width,
	const cv::cuda::PtrStepSzf& grad, const cv::cuda::PtrStepSzb& qangle,
	float sigma,
	float* block_hists,
	int cell_size_x, int cell_size_y,
	int ncells_block_x, int ncells_block_y,
	const hipStream_t& stream)
{
	const int ncells_block = ncells_block_x * ncells_block_y;
	const int patch_side = cell_size_x / 4;
	const int patch_size = cell_size_x + (patch_side * 2);
	// �������block��������ķ������㣬���Դ浽shared memory
	const int block_patch_size = ncells_block * patch_size;
	const int threads_cell = power_2up(patch_size);
	const int threads_block = ncells_block * threads_cell;
	const int half_cell_size = cell_size_x / 2;

	// x����block����������block֮������ص���y����ͬ��
	int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) /
		block_stride_x;
	int img_block_height = (height - ncells_block_y * cell_size_y + block_stride_y) /
		block_stride_y;

	/*
	// fuction:divUp(int total, int grain)
	// return:(total + grain - 1) / grain;
	*/
	const int nblocks = max_nblocks(threads_cell, ncells_block);
	dim3 grid(cv::cuda::device::divUp(img_block_width, nblocks), img_block_height);
	dim3 threads(threads_cell * ncells_block_x, ncells_block_y, nblocks);

	// Ԥ�����˹�ռ�Window����
	float scale = 1.f / (2.f * sigma * sigma);

	int hists_size = (nbins * ncells_block * patch_size * nblocks) * sizeof(float);
	int final_hists_size = (nbins * ncells_block * nblocks) * sizeof(float);
	int smem = hists_size + final_hists_size;

	/*
	// �˺���ֻ���������˵��ã�����ʱ��������ִ�в���
	// <<<>>>��������Ǻ˺�����ִ�в��������߱���������ʱ��������˺���������˵���ں˺����е��߳��������Լ��߳��������֯��

	// ����grid���ڶ�������grid��ά�Ⱥͳߴ磬��һ��grid�ж��ٸ�block��Ϊdim3����
	// Dim3 grid(grid.x, grid.y, 1)��ʾgrid��ÿ����grid.x��block��ÿ����grid.y��block������ά��Ϊ1(Ŀǰһ���˺���ֻ��һ��grid)
	// ����grid�й���grid.x*grid.y��block������grid.x��grid.y���ֵΪ65535

	// ����threads���ڶ���һ��block��ά�Ⱥͳߴ磬��һ��block�ж��ٸ�thread��Ϊdim3����
	// Dim3 threads(threads.x, threads.y, threads.z)��ʾ����block��ÿ����threads.x��thread��ÿ����threads.y��thread���߶�Ϊthreads.z��threads.x��threads.y���ֵΪ1024��threads.z���ֵΪ62
	// һ��block�й���threads.x*threads.y*threads.z��thread

	// ����smem��һ����ѡ��������������ÿ��block���˾�̬�����shared Memory���⣬����ܶ�̬�����shared memory��С����λΪbyte������Ҫ��̬����ʱ��ֵΪ0��ʡ�Բ�д

	// ����stream��һ��hipStream_t���͵Ŀ�ѡ��������ʼֵΪ�㣬��ʾ�ú˺��������ĸ���֮�С�
	*/

	if (nblocks == 4) 
		compute_hists_kernel_many_blocks<4> << <grid, threads, smem, stream >> >(img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
	else if (nblocks == 3)
		compute_hists_kernel_many_blocks<3> << <grid, threads, smem, stream >> >(img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
	else if (nblocks == 2)
		compute_hists_kernel_many_blocks<2> << <grid, threads, smem, stream >> >(img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
	else
		compute_hists_kernel_many_blocks<1> << <grid, threads, smem, stream >> >(img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);

	cudaSafeCall(hipGetLastError());
}


//-------------------------------------------------------------
//  ͨ��L2Hys_norm(Lowe-style����ȥ��L2����)��ֱ��ͼ���й�һ��
//


// ���ٹ����ڴ�
template<int size>
__device__ float reduce_smem(float* smem, float val)
{
	unsigned int tid = threadIdx.x;
	float sum = val;
	/*
	// reduce�������ã���
	   ��һ��������Դ
	   �ڶ���������
	   ������������
	*/
	cv::cuda::device::reduce<size>(smem, sum, tid, cv::cuda::device::plus<float>());

	if (size == 32)
	{
#if __CUDA_ARCH__ >= 300
		return shfl(sum, 0);
#else
		return smem[0];
#endif
	}
	else
	{
#if __CUDA_ARCH__ >= 300
		if (threadIdx.x == 0)
			smem[0] = sum;
#endif

		__syncthreads();

		return smem[0];
	}
}


template <int nthreads, // ����һ����ֱ��ͼ���߳���
	int nblocks> // ��һ��GPU block����Ŀ�ֱ��ͼ������
	__global__ void normalize_hists_kernel_many_blocks(const int block_hist_size,
	const int img_block_width,
	float* block_hists, float threshold)
{
	if (blockIdx.x * blockDim.z + threadIdx.z >= img_block_width)
		return;

	float* hist = block_hists + (blockIdx.y * img_block_width +
		blockIdx.x * blockDim.z + threadIdx.z) *
		block_hist_size + threadIdx.x;

	__shared__ float sh_squares[nthreads * nblocks];
	float* squares = sh_squares + threadIdx.z * nthreads;

	float elem = 0.f;
	if (threadIdx.x < block_hist_size)
		elem = hist[0];

	__syncthreads(); // prevent race condition (redundant?)
	float sum = reduce_smem<nthreads>(squares, elem * elem);

	float scale = 1.0f / (::sqrtf(sum) + 0.1f * block_hist_size);
	elem = ::min(elem * scale, threshold);

	__syncthreads(); // prevent race condition
	sum = reduce_smem<nthreads>(squares, elem * elem);

	scale = 1.0f / (::sqrtf(sum) + 1e-3f);

	if (threadIdx.x < block_hist_size)
		hist[0] = elem * scale;
}


void normalize_hists(int nbins,
	int block_stride_x, int block_stride_y,
	int height, int width,
	float* block_hists,
	float threshold,
	int cell_size_x, int cell_size_y,
	int ncells_block_x, int ncells_block_y,
	const hipStream_t& stream)
{
	const int nblocks = 1;

	int block_hist_size = nbins * ncells_block_x * ncells_block_y;
	int nthreads = power_2up(block_hist_size);
	dim3 threads(nthreads, 1, nblocks);

	int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
	int img_block_height = (height - ncells_block_y * cell_size_y + block_stride_y) / block_stride_y;
	dim3 grid(cv::cuda::device::divUp(img_block_width, nblocks), img_block_height);

	if (nthreads == 32)
		normalize_hists_kernel_many_blocks<32, nblocks> << <grid, threads, 0, stream >> >(block_hist_size, img_block_width, block_hists, threshold);
	else if (nthreads == 64)
		normalize_hists_kernel_many_blocks<64, nblocks> << <grid, threads, 0, stream >> >(block_hist_size, img_block_width, block_hists, threshold);
	else if (nthreads == 128)
		normalize_hists_kernel_many_blocks<128, nblocks> << <grid, threads, 0, stream >> >(block_hist_size, img_block_width, block_hists, threshold);
	else if (nthreads == 256)
		normalize_hists_kernel_many_blocks<256, nblocks> << <grid, threads, 0, stream >> >(block_hist_size, img_block_width, block_hists, threshold);
	else if (nthreads == 512)
		normalize_hists_kernel_many_blocks<512, nblocks> << <grid, threads, 0, stream >> >(block_hist_size, img_block_width, block_hists, threshold);
	else
		CV_Error(cv::Error::StsBadArg, "normalize_hists: histogram's size is too big, try to decrease number of bins");

	cudaSafeCall(hipGetLastError());
}


//---------------------------------------------------------------------
//  Linear SVM based classification
//

// return confidence values not just positive location
template <int nthreads, // Number of threads per one histogram block
	int nblocks>  // Number of histogram block processed by single GPU thread block
	__global__ void compute_confidence_hists_kernel_many_blocks(const int img_win_width, const int img_block_width,
	const int win_block_stride_x, const int win_block_stride_y,
	const float* block_hists, const float* coefs,
	float free_coef, float threshold, float* confidences)
{
	const int win_x = threadIdx.z;
	if (blockIdx.x * blockDim.z + win_x >= img_win_width)
		return;

	const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
		blockIdx.x * win_block_stride_x * blockDim.z + win_x) *
		cblock_hist_size;

	float product = 0.f;
	for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
	{
		int offset_y = i / cdescr_width;
		int offset_x = i - offset_y * cdescr_width;
		product += coefs[i] * hist[offset_y * img_block_width * cblock_hist_size + offset_x];
	}

	__shared__ float products[nthreads * nblocks];

	const int tid = threadIdx.z * nthreads + threadIdx.x;

	cv::cuda::device::reduce<nthreads>(products, product, tid, cv::cuda::device::plus<float>());

	if (threadIdx.x == 0)
		confidences[blockIdx.y * img_win_width + blockIdx.x * blockDim.z + win_x] = product + free_coef;

}

void compute_confidence_hists(int win_height, int win_width, int block_stride_y, int block_stride_x,
	int win_stride_y, int win_stride_x, int height, int width, float* block_hists,
	float* coefs, float free_coef, float threshold, int cell_size_x, int ncells_block_x, float *confidences)
{
	const int nthreads = 256;
	const int nblocks = 1;

	int win_block_stride_x = win_stride_x / block_stride_x;
	int win_block_stride_y = win_stride_y / block_stride_y;
	int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
	int img_win_height = (height - win_height + win_stride_y) / win_stride_y;

	dim3 threads(nthreads, 1, nblocks);
	dim3 grid(cv::cuda::device::divUp(img_win_width, nblocks), img_win_height);

	cudaSafeCall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_confidence_hists_kernel_many_blocks<nthreads), nblocks>,
		hipFuncCachePreferL1));

	int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) /
		block_stride_x;
	compute_confidence_hists_kernel_many_blocks<nthreads, nblocks> << <grid, threads >> >(
		img_win_width, img_block_width, win_block_stride_x, win_block_stride_y,
		block_hists, coefs, free_coef, threshold, confidences);
	cudaSafeCall(hipDeviceSynchronize());
}



template <int nthreads, // Number of threads per one histogram block
	int nblocks>  // Number of histogram block processed by single GPU thread block
	__global__ void classify_hists_kernel_many_blocks(const int img_win_width, const int img_block_width,
	const int win_block_stride_x, const int win_block_stride_y,
	const float* block_hists, const float* coefs,
	float free_coef, float threshold, unsigned char* labels)
{
	const int win_x = threadIdx.z;
	if (blockIdx.x * blockDim.z + win_x >= img_win_width)
		return;

	const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
		blockIdx.x * win_block_stride_x * blockDim.z + win_x) *
		cblock_hist_size;

	float product = 0.f;
	for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
	{
		int offset_y = i / cdescr_width;
		int offset_x = i - offset_y * cdescr_width;
		product += coefs[i] * hist[offset_y * img_block_width * cblock_hist_size + offset_x];
	}

	__shared__ float products[nthreads * nblocks];

	const int tid = threadIdx.z * nthreads + threadIdx.x;

	cv::cuda::device::reduce<nthreads>(products, product, tid, cv::cuda::device::plus<float>());

	if (threadIdx.x == 0)
		labels[blockIdx.y * img_win_width + blockIdx.x * blockDim.z + win_x] = (product + free_coef >= threshold);
}


void classify_hists(int win_height, int win_width, int block_stride_y, int block_stride_x,
	int win_stride_y, int win_stride_x, int height, int width, float* block_hists,
	float* coefs, float free_coef, float threshold, int cell_size_x, int ncells_block_x, unsigned char* labels)
{
	const int nthreads = 256;
	const int nblocks = 1;

	int win_block_stride_x = win_stride_x / block_stride_x;
	int win_block_stride_y = win_stride_y / block_stride_y;
	int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
	int img_win_height = (height - win_height + win_stride_y) / win_stride_y;

	dim3 threads(nthreads, 1, nblocks);
	dim3 grid(cv::cuda::device::divUp(img_win_width, nblocks), img_win_height);

	cudaSafeCall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(classify_hists_kernel_many_blocks<nthreads), nblocks>, hipFuncCachePreferL1));

	int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
	classify_hists_kernel_many_blocks<nthreads, nblocks> << <grid, threads >> >(
		img_win_width, img_block_width, win_block_stride_x, win_block_stride_y,
		block_hists, coefs, free_coef, threshold, labels);
	cudaSafeCall(hipGetLastError());

	cudaSafeCall(hipDeviceSynchronize());
}

//----------------------------------------------------------------------------
// Extract descriptors


template <int nthreads>
__global__ void extract_descrs_by_rows_kernel(const int img_block_width,
	const int win_block_stride_x, const int win_block_stride_y,
	const float* block_hists,
	cv::cuda::PtrStepf descriptors)
{
	// Get left top corner of the window in src
	const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
		blockIdx.x * win_block_stride_x) * cblock_hist_size;

	// Get left top corner of the window in dst
	float* descriptor = descriptors.ptr(blockIdx.y * gridDim.x + blockIdx.x);

	// Copy elements from src to dst
	for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
	{
		int offset_y = i / cdescr_width;
		int offset_x = i - offset_y * cdescr_width;
		descriptor[i] = hist[offset_y * img_block_width * cblock_hist_size + offset_x];
	}
}


void extract_descrs_by_rows(int win_height, int win_width,
	int block_stride_y, int block_stride_x,
	int win_stride_y, int win_stride_x,
	int height, int width,
	float* block_hists, int cell_size_x,
	int ncells_block_x,
	cv::cuda::PtrStepSzf descriptors,
	const hipStream_t& stream)
{
	const int nthreads = 256;

	int win_block_stride_x = win_stride_x / block_stride_x;
	int win_block_stride_y = win_stride_y / block_stride_y;
	int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
	int img_win_height = (height - win_height + win_stride_y) / win_stride_y;
	dim3 threads(nthreads, 1);
	dim3 grid(img_win_width, img_win_height);

	int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
	extract_descrs_by_rows_kernel<nthreads> << <grid, threads, 0, stream >> >(img_block_width, win_block_stride_x, win_block_stride_y, block_hists, descriptors);

	cudaSafeCall(hipGetLastError());
}


template <int nthreads>
__global__ void extract_descrs_by_cols_kernel(const int img_block_width,
	const int win_block_stride_x, const int win_block_stride_y,
	const float* block_hists,
	cv::cuda::PtrStepf descriptors)
{
	// Get left top corner of the window in src
	const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
		blockIdx.x * win_block_stride_x) * cblock_hist_size;

	// Get left top corner of the window in dst
	float* descriptor = descriptors.ptr(blockIdx.y * gridDim.x + blockIdx.x);

	// Copy elements from src to dst
	for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
	{
		int block_idx = i / cblock_hist_size;
		int idx_in_block = i - block_idx * cblock_hist_size;

		int y = block_idx / cnblocks_win_x;
		int x = block_idx - y * cnblocks_win_x;

		descriptor[(x * cnblocks_win_y + y) * cblock_hist_size + idx_in_block]
			= hist[(y * img_block_width + x) * cblock_hist_size + idx_in_block];
	}
}


void extract_descrs_by_cols(int win_height, int win_width,
	int block_stride_y, int block_stride_x,
	int win_stride_y, int win_stride_x,
	int height, int width,
	float* block_hists,
	int cell_size_x, int ncells_block_x,
	cv::cuda::PtrStepSzf descriptors,
	const hipStream_t& stream)
{
	const int nthreads = 256;

	int win_block_stride_x = win_stride_x / block_stride_x;
	int win_block_stride_y = win_stride_y / block_stride_y;
	int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
	int img_win_height = (height - win_height + win_stride_y) / win_stride_y;
	dim3 threads(nthreads, 1);
	dim3 grid(img_win_width, img_win_height);

	int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
	extract_descrs_by_cols_kernel<nthreads> << <grid, threads, 0, stream >> >(img_block_width, win_block_stride_x, win_block_stride_y, block_hists, descriptors);

	cudaSafeCall(hipGetLastError());
}

//----------------------------------------------------------------------------
// Gradients computation


template <int nthreads, int correct_gamma>
__global__ void compute_gradients_8UC4_kernel(int height, int width, const cv::cuda::PtrStepb img,
	float angle_scale, cv::cuda::PtrStepf grad, cv::cuda::PtrStepb qangle)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	const uchar4* row = (const uchar4*)img.ptr(blockIdx.y);

	__shared__ float sh_row[(nthreads + 2) * 3];

	uchar4 val;
	if (x < width)
		val = row[x];
	else
		val = row[width - 2];

	sh_row[threadIdx.x + 1] = val.x;
	sh_row[threadIdx.x + 1 + (nthreads + 2)] = val.y;
	sh_row[threadIdx.x + 1 + 2 * (nthreads + 2)] = val.z;

	if (threadIdx.x == 0)
	{
		val = row[::max(x - 1, 1)];
		sh_row[0] = val.x;
		sh_row[(nthreads + 2)] = val.y;
		sh_row[2 * (nthreads + 2)] = val.z;
	}

	if (threadIdx.x == blockDim.x - 1)
	{
		val = row[::min(x + 1, width - 2)];
		sh_row[blockDim.x + 1] = val.x;
		sh_row[blockDim.x + 1 + (nthreads + 2)] = val.y;
		sh_row[blockDim.x + 1 + 2 * (nthreads + 2)] = val.z;
	}

	__syncthreads();
	if (x < width)
	{
		float3 a, b;

		b.x = sh_row[threadIdx.x + 2];
		b.y = sh_row[threadIdx.x + 2 + (nthreads + 2)];
		b.z = sh_row[threadIdx.x + 2 + 2 * (nthreads + 2)];
		a.x = sh_row[threadIdx.x];
		a.y = sh_row[threadIdx.x + (nthreads + 2)];
		a.z = sh_row[threadIdx.x + 2 * (nthreads + 2)];

		float3 dx;
		if (correct_gamma)
			dx = make_float3(::sqrtf(b.x) - ::sqrtf(a.x), ::sqrtf(b.y) - ::sqrtf(a.y), ::sqrtf(b.z) - ::sqrtf(a.z));
		else
			dx = make_float3(b.x - a.x, b.y - a.y, b.z - a.z);

		float3 dy = make_float3(0.f, 0.f, 0.f);

		if (blockIdx.y > 0 && blockIdx.y < height - 1)
		{
			val = ((const uchar4*)img.ptr(blockIdx.y - 1))[x];
			a = make_float3(val.x, val.y, val.z);

			val = ((const uchar4*)img.ptr(blockIdx.y + 1))[x];
			b = make_float3(val.x, val.y, val.z);

			if (correct_gamma)
				dy = make_float3(::sqrtf(b.x) - ::sqrtf(a.x), ::sqrtf(b.y) - ::sqrtf(a.y), ::sqrtf(b.z) - ::sqrtf(a.z));
			else
				dy = make_float3(b.x - a.x, b.y - a.y, b.z - a.z);
		}

		float best_dx = dx.x;
		float best_dy = dy.x;

		float mag0 = dx.x * dx.x + dy.x * dy.x;
		float mag1 = dx.y * dx.y + dy.y * dy.y;
		if (mag0 < mag1)
		{
			best_dx = dx.y;
			best_dy = dy.y;
			mag0 = mag1;
		}

		mag1 = dx.z * dx.z + dy.z * dy.z;
		if (mag0 < mag1)
		{
			best_dx = dx.z;
			best_dy = dy.z;
			mag0 = mag1;
		}

		mag0 = ::sqrtf(mag0);

		float ang = (::atan2f(best_dy, best_dx) + CV_PI_F) * angle_scale - 0.5f;
		int hidx = (int)::floorf(ang);
		ang -= hidx;
		hidx = (hidx + cnbins) % cnbins;

		((uchar2*)qangle.ptr(blockIdx.y))[x] = make_uchar2(hidx, (hidx + 1) % cnbins);
		((float2*)grad.ptr(blockIdx.y))[x] = make_float2(mag0 * (1.f - ang), mag0 * ang);
	}
}


void compute_gradients_8UC4(int nbins,
	int height, int width, const cv::cuda::PtrStepSzb& img,
	float angle_scale,
	cv::cuda::PtrStepSzf grad, cv::cuda::PtrStepSzb qangle,
	bool correct_gamma,
	const hipStream_t& stream)
{
	(void)nbins;
	const int nthreads = 256;

	dim3 bdim(nthreads, 1);
	dim3 gdim(cv::cuda::device::divUp(width, bdim.x), cv::cuda::device::divUp(height, bdim.y));

	if (correct_gamma)
		compute_gradients_8UC4_kernel<nthreads, 1> << <gdim, bdim, 0, stream >> >(height, width, img, angle_scale, grad, qangle);
	else
		compute_gradients_8UC4_kernel<nthreads, 0> << <gdim, bdim, 0, stream >> >(height, width, img, angle_scale, grad, qangle);

	cudaSafeCall(hipGetLastError());
}

template <int nthreads, int correct_gamma>
__global__ void compute_gradients_8UC1_kernel(int height, int width, const cv::cuda::PtrStepb img,
	float angle_scale, cv::cuda::PtrStepf grad, cv::cuda::PtrStepb qangle)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	const unsigned char* row = (const unsigned char*)img.ptr(blockIdx.y);

	__shared__ float sh_row[nthreads + 2];

	if (x < width)
		sh_row[threadIdx.x + 1] = row[x];
	else
		sh_row[threadIdx.x + 1] = row[width - 2];

	if (threadIdx.x == 0)
		sh_row[0] = row[::max(x - 1, 1)];

	if (threadIdx.x == blockDim.x - 1)
		sh_row[blockDim.x + 1] = row[::min(x + 1, width - 2)];

	__syncthreads();
	if (x < width)
	{
		float dx;

		if (correct_gamma)
			dx = ::sqrtf(sh_row[threadIdx.x + 2]) - ::sqrtf(sh_row[threadIdx.x]);
		else
			dx = sh_row[threadIdx.x + 2] - sh_row[threadIdx.x];

		float dy = 0.f;
		if (blockIdx.y > 0 && blockIdx.y < height - 1)
		{
			float a = ((const unsigned char*)img.ptr(blockIdx.y + 1))[x];
			float b = ((const unsigned char*)img.ptr(blockIdx.y - 1))[x];
			if (correct_gamma)
				dy = ::sqrtf(a) - ::sqrtf(b);
			else
				dy = a - b;
		}
		float mag = ::sqrtf(dx * dx + dy * dy);

		float ang = (::atan2f(dy, dx) + CV_PI_F) * angle_scale - 0.5f;
		int hidx = (int)::floorf(ang);
		ang -= hidx;
		hidx = (hidx + cnbins) % cnbins;

		((uchar2*)qangle.ptr(blockIdx.y))[x] = make_uchar2(hidx, (hidx + 1) % cnbins);
		((float2*)grad.ptr(blockIdx.y))[x] = make_float2(mag * (1.f - ang), mag * ang);
	}
}


void compute_gradients_8UC1(int nbins,
	int height, int width, const cv::cuda::PtrStepSzb& img,
	float angle_scale,
	cv::cuda::PtrStepSzf grad, cv::cuda::PtrStepSzb qangle,
	bool correct_gamma,
	const hipStream_t& stream)
{
	(void)nbins;
	const int nthreads = 256;

	dim3 bdim(nthreads, 1);
	dim3 gdim(cv::cuda::device::divUp(width, bdim.x), cv::cuda::device::divUp(height, bdim.y));

	if (correct_gamma)
		compute_gradients_8UC1_kernel<nthreads, 1> << <gdim, bdim, 0, stream >> >(height, width, img, angle_scale, grad, qangle);
	else
		compute_gradients_8UC1_kernel<nthreads, 0> << <gdim, bdim, 0, stream >> >(height, width, img, angle_scale, grad, qangle);

	cudaSafeCall(hipGetLastError());
}



//-------------------------------------------------------------------
// ��һ��

texture<uchar4, 2, hipReadModeNormalizedFloat> resize8UC4_tex;
texture<uchar, 2, hipReadModeNormalizedFloat> resize8UC1_tex;

__global__ void resize_for_hog_kernel(float sx, float sy, cv::cuda::PtrStepSz<uchar> dst, int colOfs)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < dst.cols && y < dst.rows)
		dst.ptr(y)[x] = tex2D(resize8UC1_tex, x * sx + colOfs, y * sy) * 255;
}

__global__ void resize_for_hog_kernel(float sx, float sy, cv::cuda::PtrStepSz<uchar4> dst, int colOfs)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < dst.cols && y < dst.rows)
	{
		float4 val = tex2D(resize8UC4_tex, x * sx + colOfs, y * sy);
		dst.ptr(y)[x] = make_uchar4(val.x * 255, val.y * 255, val.z * 255, val.w * 255);
	}
}

template<class T, class TEX>
static void resize_for_hog(const cv::cuda::PtrStepSzb& src, cv::cuda::PtrStepSzb dst, TEX& tex)
{
	tex.filterMode = hipFilterModeLinear;

	size_t texOfs = 0;
	int colOfs = 0;

	hipChannelFormatDesc desc = hipCreateChannelDesc<T>();
	cudaSafeCall(hipBindTexture2D(&texOfs, tex, src.data, desc, src.cols, src.rows, src.step));

	if (texOfs != 0)
	{
		colOfs = static_cast<int>(texOfs / sizeof(T));
		cudaSafeCall(hipUnbindTexture(tex));
		cudaSafeCall(hipBindTexture2D(&texOfs, tex, src.data, desc, src.cols, src.rows, src.step));
	}

	dim3 threads(32, 8);
	dim3 grid(cv::cuda::device::divUp(dst.cols, threads.x), cv::cuda::device::divUp(dst.rows, threads.y));

	float sx = static_cast<float>(src.cols) / dst.cols;
	float sy = static_cast<float>(src.rows) / dst.rows;

	resize_for_hog_kernel << <grid, threads >> >(sx, sy, (cv::cuda::PtrStepSz<T>)dst, colOfs);
	cudaSafeCall(hipGetLastError());

	cudaSafeCall(hipDeviceSynchronize());

	cudaSafeCall(hipUnbindTexture(tex));
}

void resize_8UC1(const cv::cuda::PtrStepSzb& src, cv::cuda::PtrStepSzb dst) { resize_for_hog<uchar>(src, dst, resize8UC1_tex); }
void resize_8UC4(const cv::cuda::PtrStepSzb& src, cv::cuda::PtrStepSzb dst) { resize_for_hog<uchar4>(src, dst, resize8UC4_tex); }





	